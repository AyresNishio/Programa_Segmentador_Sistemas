#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

#include<omp.h>
#include<math.h>


//#include "AnalisCrit.h"

//#pragma warning(disable:4996)

#include<string>
#include <fstream>
#include<time.h>
using namespace std;

#define min(a, b) (((a) < (b)) ? (a) : (b))


//Parametros da GPU
#define threadspblock 1024

//Arquivos de entrada
string nomeEntrada = "Caso.txt";

//Arquivos de Saida
string nomeSaida = "Saida.csv";


//Parametros do Sitema
// int nbar;
// int nmed;
// int kmax;
int solSize = 10000;

//Estutura da matriz de resultados de combinações (Cn)
string nomeCnfile = "combs1000em5.txt";
const int rowsCn=1000;
const int colsCn=5;

//Variaveis de percentual
bool p25 = 0;
bool p50 = 0;
bool p75 = 0;


__device__ int critics=0; //  versao de nSols na GPU

//Etapas 
//Enumera Combinacoes 
__global__ void enumerar(int* combs, int card, long long int combsIdini, long long int combsInWave, long long int* Cn, int kmax, int nmed);

//Encontra Combinacoes criticas
__global__ void findcrit(double *E, int *combs, int *isCrit,int card, long long int combsInwave,int kmax, int nmed);

//Confirma Combinacoes criticas
__global__ void removcrit(int *combs, int* conjSol, int* isCrit, int sol, long long int combsInwave,int card, int kmax, int nmed);

//Atualiza Conjunto solucao
__global__ void preScan(int * confirmadosOut,int* confirmados, int* sum, long long int combsInWave);

__global__ void preScanSum(int* confirmados, long long int combsInWave);

__global__ void addScan(int *Aux,int *SUMS, long long int combsInWave);

__global__ void compact(int *combs, int *conjSol, int *isCrit,long long int combsInWave, int  *scaned, int kmax, int nmed);

//// Etapas da Analise de criticalidade
__device__ void d_covarAux(double *Ei, double* E, int* combs, int card, int indc, int kmax, int nmed);

__device__ bool d_isinvertible(double *mat, int m);

__global__ void scan(int *g_odata, int *g_idata, int n, int *sums);
__global__ void scanSum( int *g_idata, int n);




int main()
{
    
    //int nbar = 300;
    int nmed;
    int kmax = 4;

    // Variaveis iniciais
    const long long int wave_size = (int)pow(2, 20);


    //SolStack
    int nSols=0;
    int* Sols;

  
    // Combinacoes pre calculadas (Cn)
    long long int Cn[(rowsCn + 1) * (colsCn + 1)] = { 0 };
    ifstream Cnfile(nomeCnfile.c_str());
	for (int i = 0; i < rowsCn+1; i++)
	{
		for (int j = 0; j < colsCn+1; j++)
		{
			Cnfile >> Cn[i * (colsCn +1) + j];
            //printf("%lld ",Cn[i * (colsCn +1) + j]);
		}
        //printf("\n");
	}
	Cnfile.close();

    //Entrada programa
    ifstream arqEntrada(nomeEntrada);
    //arqEntrada >> nbar;
    arqEntrada >> nmed;
    //arqEntrada >> kmax;

    
    int* med_plan;
    med_plan = (int*)malloc(nmed * 7 *sizeof(int));
    for (int i = 0; i < nmed; i++)
	{
		for (int j = 0; j < 7; j++)
		{
			arqEntrada >> med_plan[i * 7 + j];
            //printf("%d ",med_plan[i * 7 + j]);
		}
        //printf("\n");

	}

    // Matriz Covariancia E
    double* E;
    E = (double*)malloc(nmed * nmed * sizeof(double));
    //lerE(E, nomeEfile, nmed);
    for (int i = 0; i < nmed; i++)
	{
		for (int j = 0; j < nmed; j++)
		{
			arqEntrada >> E[i * nmed + j];
            //printf("%f ",E[i * nmed + j]);
		}
        //printf("\n");
	}

    // Conjunto Solucao
    Sols = (int*)malloc(kmax * solSize * sizeof(int));

    //-----------Alocalcoes na GPU-------------------------------------------------------------
    int* combs; //Combinacoes enumeradas por ondas
    combs = (int*)malloc((size_t)wave_size * kmax * sizeof(int));

    int* isCrit;// Vetor booleano  1: Combinacao critica 0: Combinacao nao 
    isCrit = (int*)malloc(wave_size * sizeof(int));

    //Alocacoes na GPU
	double *d_E;// Matrix de Covariancia
	hipMalloc(&d_E,nmed*nmed * sizeof(double));
	hipMemcpy(d_E,E,nmed*nmed * sizeof(double),hipMemcpyHostToDevice);

	int *d_combs;//Matriz com combinacoes enumeradas
    hipMalloc(&d_combs,wave_size*kmax * sizeof(int));


	int *d_isCrit;//Matris que indica combinacoes criticas
	hipMalloc(&d_isCrit,wave_size* sizeof(int));
	//hipMemset( d_isCrit,0,wave_size* sizeof(int));

	int *d_conjSol;//conjunto solucao
    hipMalloc(&d_conjSol,solSize*kmax*sizeof(int));
    //hipMemset((void*)&d_conjSol,1,solSize*nmed*sizeof(int));
    
    long long int *d_Cn;//Combinacoes pré calculadas
    hipMalloc(&d_Cn,(rowsCn+1)*(colsCn+1)*sizeof( long long int));
    hipMemcpy(d_Cn,Cn,(rowsCn+1)*(colsCn+1)* sizeof(long long int),hipMemcpyHostToDevice);

    int* d_SUMS;//Somatórios dos blocos Scan 
    hipMalloc(&d_SUMS, (wave_size/threadspblock)*sizeof(int));
    
    int* d_Scaned;//Somatórios do Scan 
    hipMalloc(&d_Scaned, (wave_size)*sizeof(int));

    for (int card = 1; card <=kmax ;card++)
        {
            cout << "cardinalidade " << card;
            cout << ": iniciado...\n";
            long long int totalwaves = 0; // Combinacoes vizitadas em todas as ondas
            //int combsId = 0;    // Identificador de combinacoes :  (1 = 0011) (2 = 0101) ... (6 = 1100) 
            
            while (totalwaves < Cn[nmed * (colsCn + 1) + card])
            {
                long long int combsInWave = min(wave_size, Cn[nmed * (colsCn + 1) + card] - totalwaves); // Combinacoes Visitada em onda
                
                //1-Enumeracao---------------------------
                enumerar<<<wave_size/threadspblock,threadspblock>>>(d_combs, card,totalwaves+1, combsInWave, d_Cn,kmax,nmed);
                
                //2-Procura-------------------------------
                findcrit<<<wave_size/threadspblock,threadspblock>>>(d_E, d_combs, d_isCrit,card,combsInWave,kmax,nmed);

                for (int sol = 0; sol < nSols; sol++) 
                {
                    removcrit<<<wave_size/threadspblock,threadspblock>>>( d_combs, d_conjSol, d_isCrit, sol, combsInWave,card,kmax,nmed);
                }

                //4-Atualizacao do Conjunto Solucao-------
                scan<<<wave_size/threadspblock,threadspblock>>>(d_Scaned,d_isCrit,threadspblock,d_SUMS);
                scanSum<<<1,threadspblock>>>(d_SUMS,threadspblock);
                addScan<<<wave_size/threadspblock,threadspblock>>>(d_Scaned,d_SUMS, combsInWave); 
                compact<<<wave_size/threadspblock,threadspblock>>>(d_combs,d_conjSol, d_isCrit, combsInWave, d_Scaned, kmax, nmed);
            
                totalwaves += wave_size;
                //Printa percentuais para acompanhar andamento da analise de criticalidades
                if (totalwaves > Cn[nmed * (colsCn + 1) + card]/4 && !p25)
                {
                    printf("25%% ->");
                    p25 = 1;
                }
                if (totalwaves > Cn[nmed * (colsCn + 1) + card] / 2 && !p50)
                {
                    printf("50%% ->");
                    p50 = 1;
                }
                if (totalwaves > Cn[nmed * (colsCn + 1) + card] * 3 / 4 && !p75)
                {
                    printf("75%%");
                    p75 = 1;
                }

        }
        printf(" Finalizado\n");
        hipMemcpyFromSymbol(&nSols, HIP_SYMBOL(critics), sizeof(int), 0, hipMemcpyDeviceToHost);
        //Reseta cardinalidades para proxima cardinalidade
        p25 = 0; p50 = 0; p75 = 0;
        //cout <<"\ncrit found "<<conjSol.nSols<< " finalizado.\n" << endl;
        

    }

    hipMemcpy(Sols,d_conjSol,solSize*kmax* sizeof(int),hipMemcpyDeviceToHost);
    free(combs);
    free(isCrit);

    //Print Resultados ----------------------------------------------------------------------------------------------
    printf("RESULTADOS:\n");
    printf("Conjunto solucao: %i\n", nSols);
    int ncrits[colsCn];
    for (int i = 0; i < kmax; i++)
        ncrits[i] = 0;
    int card;
    for (int i = 0; i < nSols; i++) {
        card = 0; 
        for (int j = 0; j < kmax; j++) {
            if(Sols[i * kmax + j] !=-1) card++;
        }
        ncrits[card - 1] += 1;
    }
    printf("Cardinalidade -> numero de tuplas criticas\n");
    for (int i = 0; i < kmax; i++)
        printf("%i -> %i\n", i + 1, ncrits[i]);




    printf("total de tuplas criticas : %i\n", nSols);

    //Print Tempos e CSV----------------------------------------------------------------------------------------------

    //asctime(localtime(&timetoday))+
    FILE* Output_file;
    Output_file = fopen(nomeSaida.c_str(), "w");
    fprintf(Output_file ,"Cardinalidade;numero de Cks\n");
    for (int i = 0; i < kmax; i++) {
        fprintf(Output_file ,"%i;%i\n", i + 1, ncrits[i]);
    }
    fprintf(Output_file ," total de tuplas criticas : %i\n", nSols + 1);

    fprintf(Output_file, "numero de combinacoes analisadas por cardinalidade\n");
    for (int i = 1; i <= kmax; i++)
        fprintf(Output_file, "%i; %lld\n", i, Cn[nmed * (colsCn + 1) + i]);

    for (int i =0; i < nSols; i++)
    {
        int k = 0;
        for (int j =0; j < kmax; j++)
        {
            int medida = Sols[i* kmax + j];
            if(medida != -1)
            {
                if(med_plan[medida*7 + 4] == 1) fprintf(Output_file,"F(%i-%i) ",med_plan[medida*7 + 1],med_plan[medida*7 + 2]);
                if(med_plan[medida*7 + 4] == 2) fprintf(Output_file,"I(%i) ",med_plan[medida*7 + 2]);
                if(med_plan[medida*7 + 4] == 3) fprintf(Output_file,"A(%i) ",med_plan[medida*7 + 2]);
            }
            else
            {
                k++;
            }
            
        }
        fprintf(Output_file,";");
        for (int j =0; j < kmax; j++)
        {
            int medida = Sols[i* kmax + j];
            if(medida != -1) fprintf(Output_file,"%i;",med_plan[medida*7]);
            else fprintf(Output_file,"-;");
            
        }
        fprintf(Output_file,"%i \n",kmax - k);
    }
    fclose(Output_file);


    hipFree(d_conjSol);
	hipFree(d_isCrit);
	hipFree(d_E);
    hipFree(d_combs);
    hipFree(d_Cn);
    hipFree(d_SUMS);
    hipFree(d_Scaned);


}

//Etapas
__global__ void enumerar(int* combs, int card, long long int combsIdinicial,long long int combsInWave, long long int* Cn,int kmax , int nmed)
{
    int linha = threadIdx.x + blockDim.x*blockIdx.x;
    if(linha<combsInWave)
    {
        int nZ = nmed - card;
        int nO = card;
        long long int n = linha + combsIdinicial;
        for (long long int i = 0; i < nmed; i++)
        {
            nZ--;
            long long int zcomb = Cn[(nmed - 1 - i) * (colsCn + 1) +  min(nZ, nO)];
            if (zcomb < n)
            {
                combs[linha * kmax + (card-nO)] = i;
                nO--;
                nZ++;
                n = n - zcomb;
            }
        }
        for(int j=card; j<kmax;j++) {
            combs[linha * kmax + j] = -1;
        }
        
    }   
    
    //printf("\n");

    
    
}
__global__ void findcrit(double *E, int *combs,int *isCrit, int card1, long long int combsInwave,int kmax,int nmed)
{
    int ind = threadIdx.x + blockDim.x*blockIdx.x;
    
    if(ind<combsInwave){
		
        double Ei[colsCn*colsCn];
    		
        d_covarAux(Ei, E, combs, card1, ind,kmax,nmed);
        
        if (!(d_isinvertible(Ei, card1))) {
            isCrit[ind] = 1;
        }
        else {
            isCrit[ind] = 0;
		}
		
		
        free(Ei);
       
    }
}
__global__ void removcrit(int *combs, int* conjSol, int* isCrit, int sol, long long int combsInwave, int card, int kmax, int nmed)
{
    int crit = threadIdx.x + blockDim.x*blockIdx.x;
   
    if(isCrit[crit]==1)
    {
        int is = 0;
        int i = 0;
        int j = 0;
        while(conjSol[sol * kmax + i]!=-1 && i<card)
        {
            for (j = 0; j < card; j++)
            {
                if (conjSol[sol * kmax + i]==combs[crit*kmax+j])
                    break;
            }


            if (j == card)
                is = 1;
            i++;
        }
        isCrit[crit] = is;
    }
}



//Analise de Criticalidade
__device__ bool d_isinvertible(double *mat, int m) 
{

	bool inv;
	double pivo = 0.;
	for (int i = 0; i < m; i++) {

        //pivotiamento 
		int indmaior = i;
		double maior = mat[i * colsCn + i];
		for (int j = i; j < m; j++)
		{
			if (abs(maior) < abs(mat[j * colsCn + i]))
			{
				maior = mat[j * colsCn + i];
				indmaior = j;
			}
		}
		for (int j = 0; j < m; j++)
		{
			double swap = mat[i * colsCn + j];
			mat[i * colsCn + j] = mat[indmaior * colsCn + j];
			mat[indmaior * colsCn + j] = swap;
		}
		
		pivo = mat[i*colsCn + i];

		if (abs(pivo) < 0.0000000001) {
			inv = 0;
			return inv;
		}

		for (int j = 0; j < m; j++) {
			mat[i*colsCn + j] = mat[i*colsCn + j] / pivo;

		}


		for (int j = 0; j < m; j++) {
			if (j != i) {
				pivo = mat[j*colsCn + i];
				for (int l = 0; l < m; l++) {
					mat[j*colsCn + l] = mat[j*colsCn + l] - pivo * mat[i*colsCn + l];
				}
			}
		}

	}

	inv = 1;
	return inv;
}
__device__ void d_covarAux(double *Ei, double* E, int* combs, int card, int indc, int kmax, int nmed) 
{

    for (int i = 0; i < card; i++)
	{
		for (int j = 0; j < card; j++)
		{
			int m = combs[indc * kmax + i];
			int n = combs[indc * kmax + j];

			
			Ei[i * colsCn + j] = E[m * nmed + n];
		}
		
	}

}

__global__ void scan(int *g_odata, int *g_idata, int n, int *sums) 
{   
    __shared__ int temp[2*threadspblock]; 
    // allocated on invocation    
    int thid = threadIdx.x; 
    int blid = blockIdx.x;  
    int pout = 0, pin = 1;   
    // Load input into shared memory.    
    // This is exclusive scan, so shift right by one    
    // and set first element to 0   
    temp[pout*n + thid] = g_idata[thid+blid*blockDim.x] ; 
    temp[pin*n + thid] = 0;//g_idata[thid+blid*blockDim.x] ;   
    __syncthreads();
        
    for (int offset = 1; offset < n; offset *= 2)   
    {     
        pout = 1 - pout; 
        // swap double buffer indices     
        pin = 1 - pout;     
        if (thid >= offset)       
            temp[pout*n+thid] = temp[pin*n+thid - offset]+temp[pin*n+thid];     
        else       
            temp[pout*n+thid] = temp[pin*n+thid];     
        
        //if(blid==1&&thid<10) printf ("%i::%i ",thid, temp[pout*n + thid]); 
        __syncthreads();
        //if(blid==1&&thid==0)printf("\n");   
    }   
        g_odata[thid+blid*blockDim.x] = temp[pout*n+thid]; 
        // write output 
    if (thid==0)
    {
        sums[blid]=temp[n-1];
    }
    
}
__global__ void scanSum( int *g_idata, int n) 
{   
    __shared__ int temps[2*threadspblock]; 
    // allocated on invocation    
    int thid = threadIdx.x; 
    int pout = 0, pin = 1;   
    // Load input into shared memory.    
    // This is exclusive scan, so shift right by one    
    // and set first element to 0   
    temps[pout*n + thid] = g_idata[thid] ; 
    temps[pin*n + thid] = 0;  
    __syncthreads();   
    for (int offset = 1; offset < n; offset *= 2)   
    {     
        pout = 1 - pout; 
        // swap double buffer indices     
        pin = 1 - pout;     
        if (thid >= offset)       
            temps[pout*n+thid] = temps[pin*n+thid - offset]+temps[pin*n+thid];     
        else       
            temps[pout*n+thid] = temps[pin*n+thid];     
        
        __syncthreads();   
    }   
        g_idata[thid] = temps[pout*n+thid]; 
       
        
}

//Atualiza Conjunto solucao 
__global__ void preScan(int * confirmadosOut,int* confirmados, int* sum, long long int combsInWave)
{
    int tid = threadIdx.x;
    int bid= blockIdx.x;
    
    int id = tid+blockDim.x*bid;
   
    __shared__ int temp[threadspblock];
    //if (id<combsInWave){
        temp[tid]=confirmados[id];
        
        __syncthreads();  

        for (int offset = 1; offset < threadspblock; offset *= 2)   
        { 
            
            if(tid>=offset)
                temp[tid]+=temp[tid-offset];
        

            __syncthreads();  
        }
        //__syncthreads(); 
        
        if(tid==0){
            //sum[0]=0;
            //temp[0]=0;
            sum[bid]=temp[threadspblock-1];
            //printf("::%i - %i\n",sum[bid],bid);
        }
        confirmadosOut[id]=temp[tid];
    //}   

}
__global__ void preScanSum(int* confirmados, long long int combsInWave)
{
    int tid = threadIdx.x;
    int bid= blockIdx.x;
    
    int id = tid+blockDim.x*bid;
   
    __shared__ int temp[threadspblock]; 
    if (id<combsInWave){
        temp[tid]=confirmados[id];
        
        __syncthreads();  

        for (int offset = 1; offset < threadspblock; offset *= 2)   
        { 
            
            if(tid>=offset)
                temp[tid]+=temp[tid-offset];
        

            __syncthreads();  
        }
        
        confirmados[id]=temp[tid];
    }   
}

__global__ void addScan(int *Aux,int *SUMS, long long int combsInWave)
{
    int thid= threadIdx.x;
    int bid = blockIdx.x;
    int id = thid + bid*blockDim.x;
    if(id<combsInWave ){
    
        Aux[id]+=critics;
        if( bid>0)
            Aux[id]+=SUMS[bid-1];
        
    }
}

__global__ void compact(int *combs, int *conjSol, int *isCrit,long long int combsInWave, int  *scaned, int kmax, int nmed)
{
    int index =threadIdx.x + blockDim.x*blockIdx.x;
    
    if(index<combsInWave)
    {
        //printf("ev=%i",isCrit[index]);
        if(isCrit[index]==1)
        {
            int idSol=scaned[index]-1;
            for(int i = 0; i<kmax;i++){
                conjSol[idSol*kmax+i]=combs[index*kmax+i];
                
            }
            atomicAdd(&critics, 1);
            
            
        }
    }
}
